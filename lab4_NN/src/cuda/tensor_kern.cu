#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include <typeinfo>

#include "../../include/tensor_kern.h"
#include "../../include/nn_exception.cuh"

template <class Number>
__global__ void addKernel(Number* d_m1, Number* d_m2, Number* d_m3, size_t size) {
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    
    if(index < size) d_m3[index] = d_m1[index] + d_m2[index];
}

template <class Number>
__host__ void addKernelWrapper(Number* m1, Number* m2, Number* m3, size_t size) {
    // Pointer of arrays.
    Number* d_m1;
    Number* d_m2;
    Number* d_m3;

    // Allocating in Device Memory.
    hipMalloc(&d_m1, size * sizeof(Number));
    hipMalloc(&d_m2, size * sizeof(Number));
    hipMalloc(&d_m3, size * sizeof(Number));
    NNException::throwIfDeviceErrorsOccurred("Cannot allocate CUDA memory for Tensor<>.");

    // Copying in Device Memory.
    hipMemcpy(d_m1, m1, size * sizeof(Number), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2, size * sizeof(Number), hipMemcpyHostToDevice);
    hipMemcpy(d_m3, m3, size * sizeof(Number), hipMemcpyHostToDevice);
    NNException::throwIfDeviceErrorsOccurred("Cannot move data from Host To Device.");

    // Calling the kernel function.
    size_t nbBlocks = size / 1024 + (size % 1024 ? 1 : 0);
    dim3 grid(nbBlocks), block(32, 32);
    addKernel<<<grid, block>>>(d_m1, d_m2, d_m3, size);
    hipDeviceSynchronize();

    // Copying Device Memory to Host Memory.
    hipMemcpy(m3, d_m3, size * sizeof(Number), hipMemcpyDeviceToHost);
    NNException::throwIfDeviceErrorsOccurred("Cannot move data from Device To Host.");

    // Freeing Device Memory.
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);
}

template <class Number>
__global__ void dotKernel(Number* d_m1, Number* d_m2, Number* d_m3, size_t resultRows, size_t resultColumns, size_t interiorColumns) {
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if (index < resultRows * resultColumns) {
        d_m3[index] = 0;
        for (int i = 0; i < interiorColumns; i++) {
            d_m3[index] += d_m1[interiorColumns * (index / resultColumns) + i] * d_m2[index % resultColumns + i * resultColumns];
        }
    }
}

template <class Number>
__host__ void dotKernelWrapper(Number* m1, Number* m2, Number* m3, size_t resultRows, size_t resultColumns, size_t interiorColumns) {
    // Pointer of arrays.
    Number* d_m1;
    Number* d_m2;
    Number* d_m3;

    // Allocating in Device Memory.
    size_t size = resultRows * resultColumns;
    hipMalloc(&d_m1, resultRows * interiorColumns * sizeof(Number));
    hipMalloc(&d_m2, interiorColumns * resultColumns * sizeof(Number));
    hipMalloc(&d_m3, size * sizeof(Number));
    NNException::throwIfDeviceErrorsOccurred("Cannot allocate CUDA memory for Tensor<>.");

    // Copying in Device Memory.
    hipMemcpy(d_m1, m1, resultRows * interiorColumns * sizeof(Number), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2, interiorColumns * resultColumns * sizeof(Number), hipMemcpyHostToDevice);
    hipMemcpy(d_m3, m3, size * sizeof(Number), hipMemcpyHostToDevice);
    NNException::throwIfDeviceErrorsOccurred("Cannot move data from Host To Device.");

    // Calling the kernel function.
    size_t nbBlocks = resultRows * resultColumns / 1024 + (size % 1024 ? 1 : 0);
    dim3 grid(nbBlocks), block(32, 32);
    dotKernel<<<grid, block>>>(d_m1, d_m2, d_m3, resultRows, resultColumns, interiorColumns);
    hipDeviceSynchronize();

    // Copying Device Memory to Host Memory.
    hipMemcpy(m3, d_m3, size * sizeof(Number), hipMemcpyDeviceToHost);
    NNException::throwIfDeviceErrorsOccurred("Cannot move data from Device To Host.");

    // Freeing Device Memory.
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);
}

template void addKernelWrapper(int* m1, int* m2, int* m3, size_t size);
template void addKernelWrapper(float* m1, float* m2, float* m3, size_t size);
template void addKernelWrapper(double* m1, double* m2, double* m3, size_t size);

template void dotKernelWrapper(int* m1, int* m2, int* m3, size_t resultRows, size_t resultColumns, size_t interiorColumns);
template void dotKernelWrapper(float* m1, float* m2, float* m3, size_t resultRows, size_t resultColumns, size_t interiorColumns);
template void dotKernelWrapper(double* m1, double* m2, double* m3, size_t resultRows, size_t resultColumns, size_t interiorColumns);
