#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include <cub/device/device_reduce.cuh>
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE  4

const int N = 64;

void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }



/**************************/
/* BLOCK REDUCTION KERNEL */
/**************************/
__global__ void sum(const float * __restrict__ indata, float * __restrict__ outdata) {

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // --- Specialize BlockReduce for type float. 
    typedef hipcub::BlockReduce<float, BLOCKSIZE> BlockReduceT; 

    // --- Allocate temporary storage in shared memory 
    __shared__ typename BlockReduceT::TempStorage temp_storage; 

    float result;
    if(tid < N) result = BlockReduceT(temp_storage).Sum(indata[tid]);

    // --- Update block reduction value
    if(threadIdx.x == 0) outdata[blockIdx.x] = result;

    return;  
}

/********/
/* MAIN */
/********/
int main() {

    // --- Allocate host side space for 
    float *h_data       = (float *)malloc(N * sizeof(float));
    float *h_result     = (float *)malloc((N / BLOCKSIZE) * sizeof(float));

    float *d_data;      gpuErrchk(hipMalloc(&d_data, N * sizeof(float)));
    float *d_result;    gpuErrchk(hipMalloc(&d_result, (N / BLOCKSIZE) * sizeof(float)));

    for (int i = 0.; i < N; i++) h_data[i] = i;
    std::cout << "input: ";
    uint S = 0;
    for(int i = 0; i < N; i++){
        
        std::cout << h_data[i] << " ";
        S += h_data[i];

    }
    std::cout << std::endl;
    std::cout << S << std::endl; S = 0;

    gpuErrchk(hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice));

    sum<<<(N / BLOCKSIZE), BLOCKSIZE>>>(d_data, d_result);
    gpuErrchk(hipMemcpy(h_result, d_result, (N / BLOCKSIZE) * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_result, d_result, (N / BLOCKSIZE) * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "output: ";
    for(int i = 0; i < (N / BLOCKSIZE); i++){
        
        std::cout << h_result[i] << " ";
        S += h_result[i];
    }
    std::cout << std::endl;
    std::cout << S << std::endl;
    

    gpuErrchk(hipFree(d_data));
    gpuErrchk(hipFree(d_result));

    return 0;
}